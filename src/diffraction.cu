#include "hip/hip_runtime.h"
#include "diffraction.h"
#include "config.h"
#include "crystal.h"
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/pair.h>
#include <thrust/extrema.h>
#include <thrust/partition.h>
#include <thrust/count.h>

__global__ void CUDA_scattering_at_k(float* real_part,float * imag_part,
				     int * atomic_number, const float * sf_cache,
				     const float * HKL_list,const float * pos,
				     const int k,const int natoms,
				     float2 beam_center, float beam_fwhm);
__device__ float cuda_ilumination_function(const float * pos, float2 beam_center, float beam_fwhm);

__global__ void CUDA_scattering_from_all_atoms(hipfftComplex * F,float * I,const int * Z,
					       const float * pos,
					       const float * HKL_list,const int hkl_size,
					       const int start_atom, const int end_atom,
					       const int natoms,const float * atomsf,
					       const float B, float2 beam_center,
					       float beam_fwhm);
__global__ 
void CUDA_spectrum_scattering_from_all_atoms(hipfftComplex * F,float * I, const int * Z,
					     const float * pos, const float * HKL_list,
					     const int hkl_size,const int start_atom,
					     const int end_atom, const int natoms,
					     const float * atomsf,const float B,
					     const float wavelength, const float bandwidth,
					     const int wavelength_samples,float2 beam_center,
					     float beam_fwhm);
#define ELEMENTS 100
static float atomsf[ELEMENTS][9] = 
#include "atomsf.cdata"

static int atomsf_initialized = 0;

static void fill_ff_tables(){
  FILE * ff;
  char line[1024];
  char * p;
  int z;
  int i;
  ff = fopen("atomsf.lib","r");
  if(!ff){
/*    fprintf(stderr,"Info: atomsf.lib not found in current directory, using internal atomsf.lib\n");*/
    return;
  }
  
  /* Init atomsf array */
  for(i = 0;i<ELEMENTS;i++){
    atomsf[i][0] = -1;
  }
  /* skip the comments */
  for(fgets(line,1024,ff);strstr(line,"AD") == line;fgets(line,1024,ff));

  /* we're at the beginning of an atom record */
  while(line[0]){
    fgets(line,1024,ff);
    /* get Z */
    sscanf(line,"%d",&z);
    if(atomsf[z][0] == -1){
      p = line+23;
      sscanf(p,"%f",&atomsf[z][8]);
      fgets(line,1024,ff);
      p = line;
      sscanf(p,"%f",&atomsf[z][0]);
      p+= 16; 
      sscanf(p,"%f",&atomsf[z][1]);
      p+= 16; 
      sscanf(p,"%f",&atomsf[z][2]);
      p+= 16; 
      sscanf(p,"%f",&atomsf[z][3]);
      fgets(line,1024,ff);
      p = line;
      sscanf(p,"%f",&atomsf[z][4]);
      p+= 16; 
      sscanf(p,"%f",&atomsf[z][5]);
      p+= 16; 
      sscanf(p,"%f",&atomsf[z][6]);
      p+= 16; 
      sscanf(p,"%f",&atomsf[z][7]);
      /* get the last line of the atom record */
      fgets(line,1024,ff);
      /* get the first line of the next atom record */
      line[0] = 0;
      fgets(line,1024,ff);      
    }else{
      /* skip record */
      fgets(line,1024,ff);      
      fgets(line,1024,ff);      
      fgets(line,1024,ff);      
      line[0] = 0;
      fgets(line,1024,ff);            
    }
  }
}



/* d should be the size of the scattering vector |H| */
/* d is in m^-1 but we need it in A^-1 so divide  by 1^10*/
static float  scatt_factor(float d,int Z,float B){
  float res = 0;
  int i;  
  d *= 1e-10;
  /* the 0.25 is there because the 's' used by the aproxumation is 'd/2' */
  for(i = 0;i<4;i++){
    res+= atomsf[Z][i]*exp(-(atomsf[Z][i+4]+B)*d*d*0.25);
  }                
  res += atomsf[Z][8]*exp(-B*d*d/0.25);
  return res;    
}

static float ilumination_function(Experiment * exper,float * pos){
  float dist2;
  float sigma;
  /* If no fwhm is defined just return 1 everywhere */
  if(!exper->beam_fwhm){
    return 1;
  }
  /* calculate distance from the center of the beam */
  dist2 = (pos[0]-exper->beam_center_x)*(pos[0]-exper->beam_center_x)+(pos[1]-exper->beam_center_y)*(pos[1]-exper->beam_center_y);
  sigma = exper->beam_fwhm/2.355;
  printf("here\n");
  return exp(-dist2/(2*sigma*sigma));
}

Diffraction_Pattern * cuda_compute_pattern_on_list(Molecule * mol, float * HKL_list, int HKL_list_size, float B,Experiment * exp,Options * opts){
#ifndef _USE_CUDA
  sp_error_fatal("Can't use cuda when not compiled for CUDA");
#else
  int timer = sp_timer_start();
  int i,j;
  float scattering_vector_length;
  float scattering_factor_cache[ELEMENTS];
  int is_element_in_molecule[ELEMENTS];
  Diffraction_Pattern * res = (Diffraction_Pattern *)malloc(sizeof(Diffraction_Pattern));
  int points_per_percent;
  float * atom_ilumination = (float *)malloc(sizeof(float)*mol->natoms);
  int threads_per_block = 64;
  int number_of_blocks = (mol->natoms+threads_per_block-1)/threads_per_block;
 
  if(!atomsf_initialized){
    fill_ff_tables();
    atomsf_initialized = 1;
  }

  res->F = (Complex *)malloc(sizeof(Complex)*HKL_list_size);
  res->ints = (float *)malloc(sizeof(float)*HKL_list_size);
  res->HKL_list = (float *)malloc(sizeof(float)*3*HKL_list_size);
  memcpy(res->HKL_list,HKL_list,sizeof(float)*3*HKL_list_size);
  res->HKL_list_size = HKL_list_size;
  float * d_real_part;
  cutilSafeCall(hipMalloc((void **)&d_real_part,sizeof(float)*mol->natoms)); 
  float * d_imag_part;
  cutilSafeCall(hipMalloc((void **)&d_imag_part,sizeof(float)*mol->natoms));
  int * d_atomic_number;
  cutilSafeCall(hipMalloc((void **)&d_atomic_number,sizeof(int)*mol->natoms));
  cutilSafeCall(hipMemcpy(d_atomic_number,mol->atomic_number,sizeof(int)*mol->natoms,hipMemcpyHostToDevice));
  float * d_atomic_pos;
  cutilSafeCall(hipMalloc((void **)&d_atomic_pos,sizeof(float)*mol->natoms*3));
  cutilSafeCall(hipMemcpy(d_atomic_pos,mol->pos,sizeof(float)*mol->natoms*3,hipMemcpyHostToDevice));
  float * d_sf_cache;
  cutilSafeCall(hipMalloc((void **)&d_sf_cache,sizeof(float)*ELEMENTS));
  float * d_HKL_list;
  cutilSafeCall(hipMalloc((void **)&d_HKL_list,sizeof(float)*3*HKL_list_size));
  cutilSafeCall(hipMemcpy(d_HKL_list,HKL_list,sizeof(float)*3*HKL_list_size,hipMemcpyHostToDevice));

  for(j = 0 ;j< ELEMENTS;j++){
    is_element_in_molecule[j] = 0;
  }
  for(j = 0 ;j< mol->natoms;j++){
    is_element_in_molecule[mol->atomic_number[j]] = 1;
    atom_ilumination[j] = ilumination_function(exp,&(mol->pos[j*3]));
  }

  points_per_percent = 1+(HKL_list_size)/100;
  for(i = 0;i<HKL_list_size;i++){
    if(i % points_per_percent == 0){
      fprintf(stderr,"%f percent done\n",(100.0*i)/(HKL_list_size));
    }
    sp_real(res->F[i]) = 0;
    sp_imag(res->F[i]) = 0;
    scattering_vector_length = sqrt(HKL_list[3*i]*HKL_list[3*i]+HKL_list[3*i+1]*HKL_list[3*i+1]+HKL_list[3*i+2]*HKL_list[3*i+2]);
    for(j = 0;j<ELEMENTS;j++){
      if(is_element_in_molecule[j]){
	scattering_factor_cache[j] = scatt_factor(scattering_vector_length,j,B);
      }
    }
    cutilSafeCall(hipMemcpy(d_sf_cache,scattering_factor_cache,sizeof(float)*ELEMENTS,hipMemcpyHostToDevice));
    float2 beam_center = {exp->beam_center_x,exp->beam_center_y};
    CUDA_scattering_at_k<<<number_of_blocks, threads_per_block>>>(d_real_part,d_imag_part,d_atomic_number,d_sf_cache,d_HKL_list,d_atomic_pos,i,mol->natoms,beam_center,exp->beam_fwhm);
    thrust::device_ptr<float> begin =  thrust::device_pointer_cast(d_real_part);
    thrust::device_ptr<float> end =  thrust::device_pointer_cast(d_real_part+mol->natoms);
    sp_real(res->F[i]) = thrust::reduce(begin, end);
    begin =  thrust::device_pointer_cast(d_imag_part);
    end =  thrust::device_pointer_cast(d_imag_part+mol->natoms);
    sp_imag(res->F[i]) = thrust::reduce(begin, end);
    res->ints[i] = sp_cabs(res->F[i])*sp_cabs(res->F[i]);
  }
  hipFree(d_real_part);
  hipFree(d_imag_part);
  hipFree(d_atomic_number);
  hipFree(d_sf_cache);
  hipFree(d_HKL_list);
  hipFree(d_atomic_pos);
  printf("%g atoms.pixel/s\n",1.0e6*HKL_list_size*mol->natoms/sp_timer_stop(timer));
  return res;  
#endif 
}


int sort_int_map(const void * a,const void * b){
  if( *(int *)a < *(int *)b){
    return -1;
  }else if( *(int *)a == *(int *)b){
    return 0;
  }else{
    return 1;
  }  
}

Diffraction_Pattern * cuda_compute_pattern_on_list2(Molecule * mol, float * HKL_list, int HKL_list_size, float B,Experiment * exp,Options * opts){
#ifndef _USE_CUDA
  sp_error_fatal("Can't use cuda when not compiled for CUDA");
#else
  int timer = sp_timer_start();
  Diffraction_Pattern * res = (Diffraction_Pattern *)malloc(sizeof(Diffraction_Pattern));
  dim3 threads_per_block(16,16);
  const int HKL_side = (sqrt(HKL_list_size)+1);
  dim3 number_of_blocks( (HKL_side+threads_per_block.x-1)/threads_per_block.x,
			 (HKL_side+threads_per_block.y-1)/threads_per_block.y );
  
  printf("Using %d blocks\n",number_of_blocks.x*number_of_blocks.y);
  if(!atomsf_initialized){
    fill_ff_tables();
    atomsf_initialized = 1;
  }

  /* sort atoms by atomic number */
  float * sorted_pos = (float *) malloc(sizeof(float)*mol->natoms*3);
  int * sorted_atomic_number = (int *) malloc(sizeof(int)*mol->natoms);
  /* on the odd indexes we keep a key corresponding to the original index
     and on the even indexes we keep the atomic number */ 
  int * sorted_map = (int *) malloc(sizeof(int)*mol->natoms*2);
  for(int i = 0;i<mol->natoms;i++){
    sorted_map[2*i] = mol->atomic_number[i];
    sorted_map[2*i+1] = i;
  }
  qsort(sorted_map,mol->natoms,sizeof(int)*2,sort_int_map);
  /* make use of the sorted keys to sort the positions also */
  for(int i = 0;i<mol->natoms;i++){
    sorted_atomic_number[i] = sorted_map[2*i];
    sorted_pos[3*i] = mol->pos[sorted_map[2*i+1]*3];
    sorted_pos[3*i+1] = mol->pos[sorted_map[2*i+1]*3+1];
    sorted_pos[3*i+2] = mol->pos[sorted_map[2*i+1]*3+2];
  }

  res->F = (Complex *)malloc(sizeof(Complex)*HKL_list_size);
  res->ints = (float *)malloc(sizeof(float)*HKL_list_size);
  res->HKL_list = (float *)malloc(sizeof(float)*3*HKL_list_size);
  memcpy(res->HKL_list,HKL_list,sizeof(float)*3*HKL_list_size);
  res->HKL_list_size = HKL_list_size;
  int * d_atomic_number;
  (hipMalloc((void **)&d_atomic_number,sizeof(int)*mol->natoms));
  cutilSafeCall(hipMemcpy(d_atomic_number,sorted_atomic_number,sizeof(int)*mol->natoms,hipMemcpyHostToDevice));
  float * d_atomic_pos;
  cutilSafeCall(hipMalloc((void **)&d_atomic_pos,sizeof(float)*mol->natoms*3));
  cutilSafeCall(hipMemcpy(d_atomic_pos,sorted_pos,sizeof(float)*mol->natoms*3,hipMemcpyHostToDevice));
  float * d_HKL_list;
  cutilSafeCall(hipMalloc((void **)&d_HKL_list,sizeof(float)*3*HKL_list_size));
  cutilSafeCall(hipMemcpy(d_HKL_list,HKL_list,sizeof(float)*3*HKL_list_size,hipMemcpyHostToDevice));
  float * d_atomsf;
  cutilSafeCall(hipMalloc((void **)&d_atomsf,sizeof(float)*9*ELEMENTS));
  cutilSafeCall(hipMemcpy(d_atomsf,atomsf,sizeof(float)*9*ELEMENTS,hipMemcpyHostToDevice));
  hipfftComplex * d_F;
  cutilSafeCall(hipMalloc((void **)&d_F,sizeof(hipfftComplex)*HKL_list_size));
  cutilSafeCall(hipMemset(d_F,0,sizeof(hipfftComplex)*HKL_list_size));
  
  float * d_I;
  cutilSafeCall(hipMalloc((void **)&d_I,sizeof(float)*HKL_list_size));
  cutilSafeCall(hipMemset(d_I,0,sizeof(float)*HKL_list_size));

  /* we have to do this in chunks so we don't block the card forever */
  const int chunk_size = 100;

  /* extra loop around the wavelengths */
  if(opts->wavelength_samples >1 && (opts->wavelength_samples & 1) == 0){
    opts->wavelength_samples++;
  }

  for(int i = 0;i<mol->natoms;i+=chunk_size){ 
    printf("%f%% done\n",(100.0*i)/mol->natoms);
    int end_atom = sp_min(i+chunk_size,mol->natoms);
    int start_atom = i;
    float2 beam_center = {exp->beam_center_x,exp->beam_center_y};
    if(exp->bandwidth == 0 || opts->wavelength_samples == 1){
      CUDA_scattering_from_all_atoms<<<number_of_blocks, threads_per_block>>>(d_F,d_I,d_atomic_number,d_atomic_pos,d_HKL_list,HKL_list_size,start_atom,end_atom,mol->natoms,d_atomsf,B,beam_center,exp->beam_fwhm);
    }else{
      CUDA_spectrum_scattering_from_all_atoms<<<number_of_blocks, threads_per_block>>>(d_F,d_I,d_atomic_number,d_atomic_pos,d_HKL_list,HKL_list_size,start_atom,end_atom,mol->natoms,d_atomsf,B,exp->wavelength,exp->bandwidth,opts->wavelength_samples,beam_center,exp->beam_fwhm);
    }
    hipDeviceSynchronize();
    sp_cuda_check_errors();
  }
  //  calculate_pattern_from_crystal_cuda(d_I,d_F,d_HKL_list, HKL_list_size, opts);
  hipMemcpy(res->F,d_F,sizeof(hipfftComplex)*HKL_list_size,hipMemcpyDeviceToHost);
  hipMemcpy(res->ints,d_I,sizeof(float)*HKL_list_size,hipMemcpyDeviceToHost);
  sp_cuda_check_errors();
  printf("%g atoms.pixel/s\n",1.0e6*HKL_list_size*mol->natoms/sp_timer_stop(timer));
  return res;  
#endif 
}

__global__ void CUDA_scattering_at_k(float* real_part,float * imag_part, int * atomic_number, 
				     const float * sf_cache, const float * HKL_list,
				     const float * pos,const int k,const int natoms,
				     const float2 beam_center,
				     const float beam_fwhm){
  const int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(!atomic_number[i]){
    return;
  }
  if(i<natoms){
    float sf = sf_cache[atomic_number[i]] *
      sqrt(cuda_ilumination_function(&pos[i*3], beam_center, beam_fwhm));
    float tmp = 2*3.14159265F*(HKL_list[3*k]*-pos[i*3]+HKL_list[3*k+1]*-pos[i*3+1]+HKL_list[3*k+2]*-pos[i*3+2]);
    real_part[i] = sf*cos(tmp);
    imag_part[i] = sf*sin(tmp);
  }
}

__global__ void CUDA_scattering_from_all_atoms(hipfftComplex * F,float * I,const int * Z,
					       const float * pos, const float * HKL_list,
					       const int hkl_size,const int start_atom,
					       const int end_atom, const int natoms,
					       const float * atomsf,const float B,
					       float2 beam_center, float beam_fwhm){
  //  const int id = blockIdx.x*blockDim.x + threadIdx.x;
  const int id = ((blockIdx.y*blockDim.y + threadIdx.y)*gridDim.x + blockIdx.x )*blockDim.x + threadIdx.x;
  if(id<hkl_size){
    int lastZ = -1;
    float sf = 0;
    float d = sqrt(HKL_list[3*id]*HKL_list[3*id]+HKL_list[3*id+1]*HKL_list[3*id+1]+HKL_list[3*id+2]*HKL_list[3*id+2]) * 1e-10F;
    const float hkl[3] = {HKL_list[3*id],HKL_list[3*id+1],HKL_list[3*id+2]};
    for(int i = start_atom;i<end_atom;i++){ 
      if(!Z[i]){
	continue;
      }
      if(lastZ != Z[i]){
	sf = 0;
	/* the 0.25 is there because the 's' used by the aproxumation is 'd/2' */
	for(int j = 0;j<4;j++){
	  sf+= atomsf[Z[i]*9+j]*exp(-(atomsf[Z[i]*9+j+4]+B)*d*d*0.25F);
	}                
	sf += atomsf[Z[i]*9+8]*exp(-B*d*d/0.25F);
	lastZ = Z[i];
      }
      float tmp = 2*3.14159265F*(hkl[0]*-pos[i*3]+hkl[1]*-pos[i*3+1]+hkl[2]*-pos[i*3+2]);      
      float ilum = sqrt(cuda_ilumination_function(&pos[i*3], beam_center, beam_fwhm));
      F[id].x += ilum*sf*cos(tmp);
      F[id].y += ilum*sf*sin(tmp);
    }
    if(end_atom == natoms){
      I[id] =  F[id].x*F[id].x + F[id].y*F[id].y;
    }
  }    
}


__device__ float cuda_ilumination_function(const float * pos, float2 beam_center, float beam_fwhm){
  float dist2;
  float sigma;
  /* If no fwhm is defined just return 1 everywhere */
  if(!beam_fwhm){
    return 1;
  }
  /* calculate distance from the center of the beam */
  dist2 = (pos[0]-beam_center.x)*(pos[0]-beam_center.x)+
    (pos[1]-beam_center.y)*(pos[1]-beam_center.y);
  sigma = beam_fwhm/2.355;
  return exp(-dist2/(2*sigma*sigma));
}

__global__ 
void CUDA_spectrum_scattering_from_all_atoms(hipfftComplex * F,float * I, const int * Z,
					     const float * pos, const float * HKL_list,
					     const int hkl_size,const int start_atom,
					     const int end_atom, const int natoms,
					     const float * atomsf,const float B,
					     const float wavelength, const float bandwidth,
					     const int wavelength_samples,
					     const float2 beam_center,
					     const float beam_fwhm){
  //  const int id = blockIdx.x*blockDim.x + threadIdx.x;
  const int id = ((blockIdx.y*blockDim.y + threadIdx.y)*gridDim.x + blockIdx.x )*blockDim.x + threadIdx.x;
  if(id<hkl_size){
    const float H = HKL_list[3*id];
    const float K = HKL_list[3*id+1];
    const float L = HKL_list[3*id+2];
    /* The 2.0 is due to the fact that the bandwidth corresponds to 2 sigma*/
    const float w_stddev = (wavelength-(wavelength*(1-bandwidth/2)))/2.0;
    const float pi = 3.14159265;
    const float one_over_sqrt_two_pi = 1/(sqrt(2*pi));
    int lastZ = -1;
    float sf = 0;
    float d = sqrt(HKL_list[3*id]*HKL_list[3*id]+HKL_list[3*id+1]*HKL_list[3*id+1]+HKL_list[3*id+2]*HKL_list[3*id+2]) * 1e-10F;
    for(int i = start_atom;i<end_atom;i++){ 
      if(!Z[i]){
	continue;
      }
      if(lastZ != Z[i]){
	sf = 0;
	/* the 0.25 is there because the 's' used by the aproxumation is 'd/2' */
	for(int j = 0;j<4;j++){
	  sf+= atomsf[Z[i]*9+j]*exp(-(atomsf[Z[i]*9+j+4]+B)*d*d*0.25F);
	}                
	sf += atomsf[Z[i]*9+8]*exp(-B*d*d/0.25F);
	lastZ = Z[i];
      }
      float ilumination = sqrt(cuda_ilumination_function(&pos[i*3], beam_center, beam_fwhm));
      float total_weight = 0;
      hipfftComplex f = {0,0};
      for(int j = 0;j < wavelength_samples;j++){
	float new_wavelength = (wavelength*(1-bandwidth/2))+
	  (((float)j/(wavelength_samples-1))*wavelength*bandwidth);
	const float h = H*wavelength/new_wavelength;
	const float k = K*wavelength/new_wavelength;
	const float l = L*wavelength/new_wavelength;
	const float std_deviations = (new_wavelength-wavelength)/w_stddev;
	const float weight = exp(-std_deviations*std_deviations/2);
	total_weight += weight;
	float tmp = 2*pi*(h*-pos[i*3]+k*-pos[i*3+1]+l*-pos[i*3+2]);      
	f.x += sf*cos(tmp)*weight*ilumination;
	f.y += sf*sin(tmp)*weight*ilumination;
      }
      f.x = f.x/total_weight;
      f.y = f.y/total_weight;
      F[id].x += f.x;
      F[id].y += f.y;
    }
    if(end_atom == natoms){
      I[id] =  F[id].x*F[id].x + F[id].y*F[id].y;
    }
  }    
}
