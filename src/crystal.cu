#include "hip/hip_runtime.h"
#include "crystal.h"
#include "hipfft/hipfft.h"


__global__ void CUDA_add_shifted_cell(hipfftComplex * cF, float * I,hipfftComplex * F, const float * HKL_list,const int hkl_size,const float dx,const float dy, const float dz);


void calculate_pattern_from_crystal_cuda(float * d_I, hipfftComplex * d_F,float * d_HKL_list, int HKL_list_size,Options * opts){
  dim3 threads_per_block(8,8);
  const int HKL_side = (sqrt(HKL_list_size)+1);
  dim3 number_of_blocks( (HKL_side+threads_per_block.x-1)/threads_per_block.x,
			 (HKL_side+threads_per_block.y-1)/threads_per_block.y );

  //  int threads_per_block = 64;
  //  int number_of_blocks = (HKL_list_size+threads_per_block-1)/threads_per_block;
  float cell[9];
  hipfftComplex * d_cF;
  cutilSafeCall(hipMalloc((void **)&d_cF,sizeof(hipfftComplex)*HKL_list_size));
  cutilSafeCall(hipMemset(d_cF,0,sizeof(hipfftComplex)*HKL_list_size));
  crystal_cell_matrix(opts,cell);
  float dr[3];
  int k =0;
  for(int a = 0;a<opts->crystal_size[0];a++){
    for(int b = 0;b<opts->crystal_size[1];b++){
      for(int c = 0;c<opts->crystal_size[2];c++){
	dr[0] = cell[0]*a+cell[3]*b+cell[6]*c;
	dr[1] = cell[1]*a+cell[4]*b+cell[7]*c;
	dr[2] = cell[2]*a+cell[5]*b+cell[8]*c;
	printf("%f%% done\n",
	       100.0*k/(opts->crystal_size[0]*opts->crystal_size[1]*opts->crystal_size[2]));
	k++;
	CUDA_add_shifted_cell<<<number_of_blocks, threads_per_block>>>(d_cF,d_I,d_F,d_HKL_list,HKL_list_size,dr[0],dr[1],dr[2]);
	hipDeviceSynchronize();
      }
    }
  }
  hipFree(d_cF);
}

__global__ void CUDA_add_shifted_cell(hipfftComplex * cF, float * I,hipfftComplex * F, const float * HKL_list,const int hkl_size,const float dx,const float dy, const float dz){
  const int i = ((blockIdx.y*blockDim.y + threadIdx.y)*gridDim.x + blockIdx.x )*blockDim.x + threadIdx.x;
  //  const int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<hkl_size){
    /* use atan2 instead of carg */
    float phi = 2*3.14159265F*(HKL_list[3*i]*-dx+HKL_list[3*i+1]*-dy+HKL_list[3*i+2]*-dz) + atan2(F[i].y,F[i].x);
    float amp = sqrt(F[i].x*F[i].x+F[i].y*F[i].y);
    cF[i].x += cos(phi)*amp;
    cF[i].y += sin(phi)*amp;
    I[i] = cF[i].x*cF[i].x+cF[i].y*cF[i].y;
  }
}
